#include "hip/hip_runtime.h"
/*
 * (C) 2003-2018 - ntop 
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 */
#define _GNU_SOURCE
#include <hip/hip_runtime.h>
#include <signal.h>
#include <sched.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <string.h>
#include <unistd.h>
#include <sys/mman.h>
#include <errno.h>
#include <sys/time.h>
#include <time.h>
#include <pthread.h>
#include <sched.h>
#include <stdio.h>
#include <stdint.h>

#include "pfring.h"
#include "pfring_zc.h"

#include "zutils.c"

#define ALARM_SLEEP 1
#define MAX_CARD_SLOTS      32768

//c++ include
#include <iostream>
#include <cstdlib>
#include <string>
#include <vector>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <bitset>
#include <cmath>
#include <iomanip>
#include <condition_variable>
#include <mutex>
#include <assert.h>
#include <immintrin.h>

using namespace std;

const uint64_t m1  = 0x5555555555555555; //binary: 0101...
const uint64_t m2  = 0x3333333333333333; //binary: 00110011..
const uint64_t m4  = 0x0f0f0f0f0f0f0f0f; //binary:  4 zeros,  4 ones ...
const uint64_t m8  = 0x00ff00ff00ff00ff; //binary:  8 zeros,  8 ones ...
const uint64_t m16 = 0x0000ffff0000ffff; //binary: 16 zeros, 16 ones ...
const uint64_t m32 = 0x00000000ffffffff; //binary: 32 zeros, 32 ones
const uint64_t hff = 0xffffffffffffffff; //binary: all ones
const uint64_t h01 = 0x0101010101010101; //the sum of 256 to the power of 0,1,2,3...

struct timeval startCPU7;
struct timeval endCPU7;
unsigned long diffCPU7=0;

struct timeval startGPU;
struct timeval endGPU;
unsigned long diffGPU=0;

struct timeval startCPU;
struct timeval endCPU;
unsigned long diffCPU=0;

struct timeval startCUDA;
struct timeval endCUDA;
unsigned long diffCUDA=0;

struct timeval startREAL;
struct timeval endREAL;
unsigned long diffREAL = 0;

struct timeval startREALGPU;
struct timeval endREALGPU;
unsigned long diffREALGPU = 0;
unsigned long long int REALtotaltime = 0;
unsigned long long int printtime = 0;
unsigned long long int REALCPU = 0;
unsigned long long int REALGPU = 0;
unsigned long long int REALcountpak[7] = {0};
float totalspeed = 0.0;
int CPUbyte[7] = {0};
int countpak[7] = {0};
int count_intermittent[7] = {0};
int totalpacket[7]={0};
int counterT2[7]={0};
int justanumber[7] = {0};
int modechoose = -1;
int totallock[7] = {0};
int totalcantlock[7] = {0};
//int numofpakcal = 100;

///////////////////////////////////////////////////////
double countrate = 1;
float AdaptiveThreashold = 0.4; //0.8
int timesthreashold = 1;
#if defined(_hpma)
bool CPUPrefilter[7]={1,1,1,1,1,1,1}; //ccc
#elif defined(_GPU)
bool CPUPrefilter[7]={0,0,0,0,0,0,0}; //ccc
#endif

//int hpmamode[21] = {3,2,2,2,2,2,2,2,2,2,2,1,1,1,1,1,1,0,0,0,0};
int hpmamode[11] = {2,1,1,1,1,1,1,1,1,0,0};
///////////////////////////////////////////////////////

bool timesupCPU = false;
bool timesupGPU = false;
int timesupcount = 0;
bool timesup = false;
bool samepak = true;
inline int popcount_3(uint64_t x)
{
    x -= (x >> 1) & m1;             //put count of each 2 bits into those 2 bits
    x = (x & m2) + ((x >> 2) & m2); //put count of each 4 bits into those 4 bits 
    x = (x + (x >> 4)) & m4;        //put count of each 8 bits into those 8 bits 
    return (x * h01)>>56;  //returns left 8 bits of x + (x<<8) + (x<<16) + (x<<24) + ... 
}

inline unsigned long long int BitArrayToInt(bool arr[], int count1, int count2)
{
    unsigned long long int ret = 0;
    unsigned long long int tmp;
    for (int i = count1; i < count2; i++) {
        tmp = arr[i];
        ret |= tmp << (count2 - i - 1);
    }
    return ret;
}

inline int BitCount ( unsigned char arr[], int count1, int count2)
{
	int ret = 0;
	for(int i=count1/8;i<count2/8;i++)
	{
		bitset<8> a(arr[i]);
		ret += a.count();
	}
	bitset<8> b(arr[count2/8] & ( 0xff << (8-(count2 & 0x07))));
	ret += b.count();

	return ret;
}

/*unsigned long long int BitArrayToInt(bool arr[], int count1, int count2)
{
	unsigned long long int ret = 0;
	int tmp;
	for(int i = count1; i < count2; i++)
	{
		if(arr[i]==true)
		{
			ret += pow(2, count2-i);
			//cout << ret << endl;
		}
	}
	return ret;
}*/

//cuda
hipError_t r;

//include mwm
#define DEBUG_AC
#include "mwm.c"
#include "acsmx.c"
#define err printf("file:%s->line(%d):%d\n", __FILE__, __LINE__, hipGetErrorString(r));

//defined load file
#define T1c_FILE_PATH "DB/T1_char"
//#define T1b_FILE_PATH "DB/T1_table_s"
#define T1b_FILE_PATH "DB/T1_check"
#define T1_FILE_PATH "DB/T1_table_sc"
#define BT_FILE_PATH "DB/BT_table_s"
#define T2_FILE_PATH "DB/T2_s"
#define T2b_FILE_PATH "DB/T2_bit"
#define T2c_FILE_PATH "DB/T2_char"

#define PAT_FILE_PATH "DB/patNumdata"
#define PATTERN_FILE_PATH "DB/patdata"
#define IF_DROP_FILE_PATH "if_drop"
//thread and semaphore
#include <csignal>
#include <semaphore.h>
sem_t os_sem;
sem_t *sem_shm;

int num_threads = 1;
int cluster_id = -1;

//correctness

//defined about GPU
unsigned char *T1, *BT, *T2;
unsigned char *T2b;
unsigned char *T1b;
unsigned char *T1c;
unsigned char *T2c;
u_char* T2ptr[256*256];
static unsigned short *H1;
unsigned int t1size=0, btsize=0, t2size=0, win_size=0, ingpubuf_pktnum=1000, Output_table_size=0, pcapfile_pktnum=0,HMA_mode=false, t2bsize=0, t1bsize=0, t1csize=0, t2csize=0;
unsigned short *gFST1D, *gfailure_table, *gCST1D,*gCST_CDF_size;
short *gfinal_state_table;
bool test_state=false, justgpu = false, justprecpu=false;

unsigned int cap_pkt=0, precount_total=0, pat1or2count_total=0;
fstream outThroughput, fcap, all_test;
fstream outputyo;
fstream outputyoGPU;
//global
int two_NIC = 0;
int match_mode = 0;
unsigned int payloadlen = 1458;
int buffershift = 1472;
unsigned int rec_pkt_size = 1500;
unsigned long long int CountNumofPacketToGpuFun1 = 0;
unsigned long long int CountNumofPacketToGpuFun2 = 0;
int *counterGpuSnort;
int *d_counterGpuSnort;
int counterGPUHit = 0;
int countGPUthreadtimes = 0;
int totalcountpak[7] = {0};
int totalbushibaRR = 0;
int a = 0;

//global memory to gpu
int buffer_size ;//=blocksNumper*threadsperBlock;
int blocksNumper;// =32;//16
int threadsperBlock ;//= 256;//512
int bytes;// = sizeof(u_char)*buffer_size*1458;
double buffer_times ;
unsigned long long int handlepak[7] = {0};
unsigned long long int temp_test[4] = {0,0,0,0};
unsigned long long int temp_whoami[7] = {0,0,0,0,0,0,0};
unsigned long long int kernel_sum = 0; //nfound_GPU[]
unsigned long long int kernel_tablehits = 0; //Tablehits[]
unsigned long long int kernel_launch = 0;
bool GPUPrefilter = false;
int *d_nfound_GPU1,*d_nfound_GPU2;
int *d_whoami_GPU;
bool *d_Tablehits1,*d_Tablehits2;
bool *Tablehits1,*Tablehits2;
int *nfound_GPU1,*nfound_GPU2;
int *whoami_GPU;
bool CPUrecord = true;
int intoGPUtime=0;
u_char* d_fullTable;
u_char* fullTable;
texture <u_char,1,hipReadModeElementType> texFulltable;
texture <u_char,1,hipReadModeElementType> texacsm;

u_char* first_buffer;
u_char* second_buffer;
u_char* __restrict__ Central_memory;
u_char* __restrict__ Central_memory_tail; // = Central_memory;
u_char* __restrict__ CPU_Central_memory;
bool DB_Check = 0;
int DB_Count = 0;
int *CPU_Count;
u_char** Buffer_location;
bool isReady[2]={true,true};
mutex lockurmother;
condition_variable cv_buffer;
condition_variable cv_buffer1;
condition_variable cv_buffer2;
int alldead = 0;
bool firsttimerecord = true;

unsigned lastdrops=0,rec_drop_total=0; //2019/1/9 drop

u_char* d_Central_memory;

static u_char bitmasks[8];

//ac
ACSM_STRUCT * acsm;

//mwm
MWM_STRUCT *ps;
vector <bool> T1_1bit;
bitset <256*256> T1_bitset;
static bool *  T1_bool;
unsigned char T1_char[256*256/8];
unsigned char *d_T1_char;
unsigned char T2_char[256*256/8];
unsigned char *d_T2_char;
unsigned char *d_BT_char;
bool T1b_bool[256*256];
bool T2_bool[256*256];
bool * T1_char1;
bool * T1_char2;

#ifdef _test
char* patArray[2000];
#endif
int stat_mwm;

void fastMemcpy(unsigned char *pvDest, unsigned char *pvSrc, size_t nBytes) {
  assert(nBytes % 32 == 0);
  assert((intptr_t(pvDest) & 31) == 0);
  assert((intptr_t(pvSrc) & 31) == 0);
  const __m256i *pSrc = reinterpret_cast<const __m256i*>(pvSrc);
  __m256i *pDest = reinterpret_cast<__m256i*>(pvDest);
  int64_t nVects = nBytes / sizeof(*pSrc);
  for (; nVects > 0; nVects--, pSrc++, pDest++) {
    const __m256i loaded = _mm256_stream_load_si256(pSrc);
    _mm256_stream_si256(pDest, loaded);
  }
  _mm_sfence();
}

//string to int, three char to one int
int StrToInt(string str)
{
	unsigned int num=0;
	stringstream tmpstrtonum;
	tmpstrtonum << str[0] << str[1] << str[2];
	tmpstrtonum >> num;
	tmpstrtonum.clear();
	return num;
}

//rule of stable_sort, large to small
bool sortRule(const string& s1, const string& s2)
{
	return s1.size() > s2.size();
}

int MatchFound (void* id, int index, void *data)
{
	//printf("%s\n",id);
	return 0;
}
__device__ void MatchFound_AC (void* id)
{
	//printf("1");
	printf ("%s\n",(char *)id);
	//return 0;
}


unsigned long GetFileLength (FILE *filename)
{
	unsigned long pos = ftell(filename);
	unsigned long len = 0;
	fseek (filename, 0 ,SEEK_END);
	len = ftell (filename);
	fseek (filename,pos,SEEK_SET);
	return len;

}

/////////////////////////pop
//#include "popcnt.cpp"
inline int popcnt_naive(unsigned *buf , int n){
	int cnt =0 ;
	unsigned v;
	do {
		v=*buf;
		while(v){
			cnt += v&1;
			v>>=1;
		}
		buf++;
	}while(--n);
	return cnt;
}

/////////20190606 for check T1_char
inline bool CheckT1(int pos){
	unsigned char idx = T1_char[pos >> 3]; // pos/8
	//string str = bitset<8>(T1_char[ pos >> 3 ]).to_string();
	int check = pos & 0x07;
	if( idx & ( 0x01 << (7-check) ) )
	{
		return true;
	}else
	{
		return false;
	}
}
////////20200623 for check gpu T1_char
__device__ bool d_CheckT1(int pos, unsigned char *T1_char){
	unsigned char idx = T1_char[pos >> 3];
	int check = pos & 0x07;
	if( idx & ( 0x01 << (7-check) ) )
	{
		return true;
	}else
	{
		return false;
	}
}

////////20190718 for check T2_char
inline bool CheckT2(int pos){
	unsigned char idx = T2_char[pos >> 3]; // pos/8
	int check = pos & 0x07;
	if( idx & (0x01 << (7-check) ) )
	{
		return true;
	}else
	{
		return false;
	}


}
////////20200623 for check gpu T2_char
__device__ bool d_CheckT2(int pos, unsigned char *T2_char){
	unsigned char idx = T2_char[pos >> 3];
	int check = pos & 0x07;
	if( idx & (0x01 << (7-check) ) )
	{
		return true;
	}else
	{
		return false;
	}
}

////////20200623 for BitCount
__device__ int d_BitCount(unsigned char *arr, int count1, int count2){
	int ret = 0;
	/*for(int i=count1/8;i<count2/8;i++)
	{
		bitset<8> a(arr[i]);
		ret += a.count();
	}
	bitset<8> b(arr[count2/8] & (0xff << (8-(count2 & 0x07))));
	ret += b.count();*/
	
	for(int i=count1/8;i<count2/8;i++)
	{
		ret += __popcll((int)arr[i]);
	}
	ret += __popcll((int)arr[count2/8] & (0xff << (8-(count2 & 0x07))));


	return ret;
}

inline static size_t popcnt(uint8_t v){
	size_t rt;
#if INTRIN_WORDSIZE>=64
	printf("if \n\n");
	rt = popcnt((uint64_t)v);
#else
	printf("else \n\n");
	rt = popcnt((uint32_t)v);
#endif
	return rt;
}


static struct timeval startTime;
u_int8_t bidirectional = 0, wait_for_packet = 1, flush_packet = 0, do_shutdown = 0, verbose = 0;

pfring_zc_cluster *zc;

struct dir_info {
  u_int64_t __padding 
  __attribute__((__aligned__(64)));

  pfring_zc_queue *inzq, *outzq;
  pfring_zc_pkt_buff *tmpbuff;

  u_int64_t numPkts;
  u_int64_t numBytes;
  
  char *in_dev;
  char *out_dev;

  int bind_core;
  pthread_t thread
  __attribute__((__aligned__(64)));
};
struct dir_info dir[32]; //dir[2]

/* ******************************** */
void print_stats() {
	int totalCPUstate = 0;
	int totalintermittent = 0;
	int totaljustanumber = 0;
	
	struct timeval endTime;
	double deltaMillisec[num_threads];
	static u_int8_t print_all;
	/*static u_int64_t lastPkts = 0;
	static u_int64_t lastBytes = 0;
	static u_int64_t lastDrops = 0;*/
	unsigned long long int lastPkts[num_threads];
	unsigned long long int lastBytes[num_threads];
	unsigned long long int lastDrops[num_threads];

	double diff[num_threads], dropsDiff[num_threads], bytesDiff[num_threads];
	static struct timeval lastTime;
	//char buf1[64], buf2[64], buf3[64];
	char buf1[num_threads][64]={0}, buf2[num_threads][64]={0}, buf3[num_threads][64]={0};
	unsigned long long nBytes = 0, nPkts = 0/*, nDrops = 0*/;
	unsigned int drop_sep[num_threads]={0};
	unsigned int nowdrops= 0; //2019/1/9 drop
	pfring_zc_stat stats;
	int i;
	
	if(startTime.tv_sec == 0) {
		gettimeofday(&startTime, NULL);
		print_all = 0;
	} else
	{
		print_all = 1;
	}
	
	gettimeofday(&endTime, NULL);
	for(i=0;i<num_threads;i++)
	{
		deltaMillisec[i] = delta_time(&endTime, &startTime);
	}

  /*for (i = 0; i < 1 + bidirectional; i++) {
    nBytes += dir[i].numBytes;
    nPkts += dir[i].numPkts;
    if (pfring_zc_stats(dir[i].inzq, &stats) == 0)
      nDrops += stats.drop;
  }*/
	/*for (i = 0; i<num_threads;i++)
	{
		nBytes += dir[i].numBytes;
		nPkts += dir[i].numPkts;
		if (pfring_zc_stats(dir[i].inzq, &stats) == 0)
		{
			nDrops += stats.drop;
		}
	}*/

  /*fprintf(stderr, "=========================\n"
	  "Absolute Stats: %s pkts (%s drops) - %s bytes\n", 
	  pfring_format_numbers((double)nPkts, buf1, sizeof(buf1), 0),
	  pfring_format_numbers((double)nDrops, buf3, sizeof(buf3), 0),
	  pfring_format_numbers((double)nBytes, buf2, sizeof(buf2), 0));

  if(print_all && (lastTime.tv_sec > 0)) {
    char buf[256];

    deltaMillisec = delta_time(&endTime, &lastTime);
    diff = nPkts-lastPkts;
    dropsDiff = nDrops-lastDrops;
    bytesDiff = nBytes - lastBytes;
    bytesDiff /= (1000*1000*1000)/8;

    snprintf(buf, sizeof(buf),
	     "Actual Stats: %s pps (%s drops) - %s Gbps",
	     pfring_format_numbers(((double)diff/(double)(deltaMillisec/1000)),  buf2, sizeof(buf2), 1),
	     pfring_format_numbers(((double)dropsDiff/(double)(deltaMillisec/1000)),  buf1, sizeof(buf1), 1),
	     pfring_format_numbers(((double)bytesDiff/(double)(deltaMillisec/1000)),  buf3, sizeof(buf3), 1));
    fprintf(stderr, "%s\n", buf);
  }
    
  fprintf(stderr, "=========================\n\n");*/
  	cout<<"=========="<<endl;
	double totalbytes = 0.0;
	double totaldiff = 0;
	double totaldropsDiff = 0;
	for(i=0;i<num_threads;i++)
	{
		if (pfring_zc_stats(dir[i].inzq, &stats) == 0)
		{
			//cout<<"recv: "<<stats.recv<<" sent: "<<stats.sent<<" drop: "<<stats.drop<<endl;
			drop_sep[i] = (unsigned int)stats.drop;
			fprintf(stderr, "Thread: %d Now: %s pkts (%s drops)  ", //- %s bytes\n
				i,
				pfring_format_numbers((double)dir[i].numPkts, buf1[i], sizeof(buf1[i]), 0),
				pfring_format_numbers((double)drop_sep[i], buf3[i], sizeof(buf3[i]), 0)
				/*pfring_format_numbers((double)dir[i].numBytes, buf2, sizeof(buf2), 0)*/);

			if(print_all && (lastTime.tv_sec > 0)) 
			{
				char buf[256];

				deltaMillisec[i] = delta_time(&endTime, &lastTime);
				diff[i] = dir[i].numPkts-lastPkts[i];
				dropsDiff[i] = drop_sep[i]-lastDrops[i];
				bytesDiff[i] = dir[i].numBytes - lastBytes[i];
			totaldiff+=diff[i];
			totaldropsDiff+=dropsDiff[i];
			totalbytes+=(double)bytesDiff[i];
				//bytesDiff[i] /= (1000*1000*1000)/8;
				bytesDiff[i] /= (1000*1000*1000)/8;
				cout<<" lastBytes: "<<lastBytes[i]<<" dir[i].numBytes: "<<dir[i].numBytes<<" ";

				snprintf(buf, sizeof(buf),
					"Throughput: %s Gbps", //ALL: %s pps (%s drops) - %s Gbps
					/*pfring_format_numbers(((double)diff[i]/(double)(deltaMillisec/1000)),  buf2[i], sizeof(buf2[i]), 1),
					pfring_format_numbers(((double)dropsDiff[i]/(double)(deltaMillisec/1000)),  buf1[i], sizeof(buf1[i]), 1),*/
					pfring_format_numbers(((double)bytesDiff[i]/countrate),  buf3[i], sizeof(buf3[i]), 1));
				fprintf(stderr, "%s", buf);
			}
			
			//fprintf(stderr, "=========================\n\n");
			cout<<endl;
			
			lastPkts[i] = dir[i].numPkts;
			lastDrops[i] = drop_sep[i];
			lastBytes[i] = dir[i].numBytes;

			lastTime.tv_sec = endTime.tv_sec, lastTime.tv_usec = endTime.tv_usec;
		}
	}
	double throughput=0;
	for(int i=0;i<num_threads;i++)
	{
			throughput += bytesDiff[i];
			nowdrops += drop_sep[i]; //20191/9 drop
	}
	rec_drop_total = nowdrops - lastdrops; //20191/9 drop
	lastdrops = nowdrops; //20191/9 drop
	cout << "if_drop  = " << rec_drop_total << ", droprate:" << totaldropsDiff << "/" << totaldiff+totaldropsDiff << " (" << totaldropsDiff/(totaldiff+totaldropsDiff) << ") " <<endl;
	cout<<"ALL Throughput = "<<(double)throughput/(double)countrate<<endl;
	totalbytes=0;

	//2019/1/14 if_drop from sigproc to printstate
	outThroughput.open( IF_DROP_FILE_PATH ,ios::out | ios::ate);
	outThroughput << rec_drop_total;
	outThroughput.close();


	cout << "rec_drop_total: " << rec_drop_total << endl;
	if(intoGPUtime!=0)
	{
		cout << "wow:" << totalbushibaRR << "/" << intoGPUtime << "= " << totalbushibaRR/intoGPUtime << endl;
	}
	//outputyoGPU << endl;
	for(int i=0;i<num_threads;i++)
	{
		totalintermittent += count_intermittent[i];
		totaljustanumber += justanumber[i];
		cout << justanumber[i] << " ";
	}
	cout << endl;
	cout << totaljustanumber << endl;
	for(int i=0;i<num_threads;i++)
	{
		if(CPUPrefilter[i]==true)
		{
			//if(countpak[i]!=0)
			//{
				//cout << "CPUPrefilter["<<i<<"]: on  speed: " << countpak[i]/countrate*1516*8/1000000000 << endl;
				cout << "CPUPrefilter["<<i<<"]: on  speed: " << totalpacket[i]/countrate*1514*8/1000000000 << endl;
				//outputyoGPU << totalpacket[i]/countrate*1514*8/1000000000 << " ";
				totalspeed += totalpacket[i]/countrate*1514*8/1000000000;
				//outputyoGPU << (counterT2[i]+count_intermittent[i])/countrate*1530*8/1000000000 << " ";
			//}else if(countpak[i]==0)
			//{
				//cout << "CPUPrefilter["<<i<<"]: on  speed: not yet!" << endl;
			//}
		}else if(CPUPrefilter[i]==false)
		{
			//if(countpak[i]!=0)
			//{
				//cout << "CPUPrefilter["<<i<<"]: off speed: " << countpak[i]/countrate*1516*8/1000000000 << endl;
				cout << "CPUPrefilter["<<i<<"]: off  speed: " << (intoGPUtime*buffer_size*(justanumber[i]/(double)totaljustanumber))/countrate*1514*8/1000000000 << endl;
				//outputyoGPU << (intoGPUtime*buffer_size*(justanumber[i]/(double)totaljustanumber))/countrate*1514*8/1000000000 << " ";
				totalspeed += (intoGPUtime*buffer_size*(justanumber[i]/(double)totaljustanumber))/countrate*1514*8/1000000000;
				//outputyoGPU << (counterT2[i]+count_intermittent[i])/countrate*1530*8/1000000000 << " ";
			//}else if(countpak[i]==0)
			//{
				//cout << "CPUPrefilter["<<i<<"]: off speed: not yet!" << endl;
			//}
		}
		//totalspeed = totalspeed + countpak[i]/countrate*1516*8/1000000000;
		countpak[i] = 0;
	}
	totalCPUstate = 0;
	for(int i=0;i<num_threads;i++)
	{
		if(CPUPrefilter[i] == true)
		{
			//outputyoGPU << " 1";
		}else if (CPUPrefilter[i] == false)
		{
			//outputyoGPU << " 0";
		}
	}
	cout << "total speed: " << totalspeed << endl;
	if(totalspeed>=5)
	{

		if(firsttimerecord == false)
		{
			outputyo << totalspeed << " ";
			/*if(CPUPrefilter[0]==0)
			{
				outputyo << totalspeed << endl;
			}else if(CPUPrefilter[0]==1)
			{
				outputyo << totalspeed << endl;
			}*/
		}
		firsttimerecord = false;
		for(int i=0;i<num_threads;i++)
		{
			outputyo << CPUPrefilter[i] << " ";
		}
		outputyo << endl;	
	}
	
	totalspeed = 0;

	/*if(DB_Check==0)
	{
		for(int i=0;i<num_threads;i++)
		{
			cout << setw(3) << CPU_Count[i] << " | ";
		}
		cout << isReady[0] << endl;
		for(int i=0;i<num_threads;i++)
		{
			cout << "  0 | ";
		}
		cout << isReady[1] << endl;
	}else if(DB_Check==1)
	{
	
		for(int i=0;i<num_threads;i++)
		{
			cout << "  0 | ";
		}
		cout << isReady[0] << endl;
		for(int i=0;i<num_threads;i++)
		{
			cout << setw(3) << CPU_Count[i+num_threads] << " | ";
		}
		cout << isReady[1] << endl;
	}*/

	cout << "AllDead: " << alldead << endl;
	alldead = 0;
	cout << "countGPUthreadtimes: " << countGPUthreadtimes << endl;
	countGPUthreadtimes = 0;

	/*cout << "Totalcantlock: ";
	for(int i=0;i<num_threads;i++)
	{
		cout << totalcantlock[i] << ":" << justanumber[i] << " _" << totalcantlock[i]+justanumber[i] << "_ " << "(" << ((double)totalcantlock[i]/(totalcantlock[i]+justanumber[i])) << ")" << " ";
	}
	cout <<endl;*/

	timesupcount++; //RRR
	double totalsps[7] = {0.0};
	double totalpak[7] = {0.0};
	double totalpercent[7] = {0.0};
	if(timesupcount >= timesthreashold)
	{
		timesup = true;
		timesupcount = 0;
		//int threadpak = intoGPUtime*buffer_size/num_threads;
		for(int i=0;i<num_threads;i++)
		{

			//CPUPrefilter[i] = !CPUPrefilter[i];

			totalsps[i] = counterT2[i] + count_intermittent[i];
			totalpak[i] = intoGPUtime*buffer_size/num_threads + totalpacket[i];

			//totalpercent[i] = (double)totalsps[i]/(double)totalpak[i];
			//cout << i << ": (" << counterT2[i] << "+" << count_intermittent[i] << ")/(" << intoGPUtime*buffer_size/num_threads << "+" << totalpacket[i] << ")= " << totalpercent[i];
			if(CPUPrefilter[i]==true)
			{
				totalpercent[i] = counterT2[i]/(double)totalpacket[i];
				cout << i << ": (" << counterT2[i] << "+" << count_intermittent[i] << ")/(" << (intoGPUtime*buffer_size*(justanumber[i]/(double)totaljustanumber)) << "+" << totalpacket[i] << ")=" << totalpercent[i] << endl;
			}else if(CPUPrefilter[i] == false)
			{
				//totalpercent[i] = count_intermittent[i]/(double)(intoGPUtime*buffer_size/num_threads);
				totalpercent[i] = count_intermittent[i]/(double)(intoGPUtime*buffer_size*(justanumber[i]/(double)totaljustanumber));
				cout << i << ": (" << counterT2[i] << "+" << count_intermittent[i] << ")/(" << (intoGPUtime*buffer_size*(justanumber[i]/(double)totaljustanumber)) << "+" << totalpacket[i] << ")=" << totalpercent[i] << endl;
			}
			if(totalpercent[i] > (AdaptiveThreashold-0.002))
			{
				if(CPUPrefilter[i] == false)
				{
					//CPUPrefilter[i] = true;
				}
			}else if(totalpercent[i] < (AdaptiveThreashold-0.002))
			{
				if(CPUPrefilter[i] == true)
				{
					//CPUPrefilter[i] = false;
				}
			}
			if(i==0 && totalpercent[0]>=0)
			{
				//modechoose = ((int)(totalpercent[0]*100)/5);
				modechoose = ((int)(totalpercent[0]*100)/10);
				int temppercent = ((int)(totalpercent[0]*100))%10;
				//if(temppercent>=4||temppercent>=9)
				if(temppercent>=9)
				{
					modechoose+=1;
				}
				//cout << "modechoose: " << modechoose << endl;
			}
			unique_lock<mutex> lk(lockurmother);
			counterT2[i] = 0;
			count_intermittent[i] = 0;
			totalpacket[i] = 0;
			totalpercent[i] = 0;
			justanumber[i] = 0;
			totallock[i] = 0;
			totalcantlock[i] = 0;
			lk.unlock();
		}
		//cout << hpmamode[modechoose] << endl;
		for(int i=0;i<num_threads;i++) //CGCLB
		{
			if(modechoose != -1)
			{
				if(i<hpmamode[modechoose])
				{
					//CPUPrefilter[i] = false;
				}else if(i>=hpmamode[modechoose])
				{
					//CPUPrefilter[i] = true;
				}
			}
		}
		intoGPUtime = 0;
		timesup = false;
		timesupGPU = true;
		totalCPUstate = 0;
		totalintermittent = 0;
		totaljustanumber = 0;
		modechoose = -1;
	}

}

/* ******************************** */

void sigproc(int sig) {
  static int called = 0;
  fprintf(stderr, "Leaving...\n");
  if(called) return; else called = 1;

  do_shutdown = 1;

  //print_stats();
  
  /*pfring_zc_queue_breakloop(dir[0].inzq);
  if (bidirectional) pfring_zc_queue_breakloop(dir[1].inzq);*/
	for(int i=0;i<num_threads;i++)
	{
		pfring_zc_queue_breakloop(dir[i].inzq);
	}
	
	/*outThroughput.open( IF_DROP_FILE_PATH ,ios::out | ios::ate);
	outThroughput << rec_drop_total;
	outThroughput.close();

	cout << "rec_drop_total: " << rec_drop_total << endl;*/
}

/* *************************************** */

void printHelp(void) {
	printf("compile success!!!\n");
  printf("zbounce - (C) 2014-2018 ntop.org\n");
  printf("Using PFRING_ZC v.%s\n", pfring_zc_version());
  printf("A packet forwarder application between interfaces.\n\n");
  printf("Usage:  zbounce -i <device> -o <device> -c <cluster id> [-b]\n"
	 "                [-h] [-g <core id>] [-f] [-v] [-a]\n\n");
  printf("-h              Print this help\n");
  printf("-i <device>     Ingress device name\n");
  printf("-o <device>     Egress device name\n");
  printf("-c <cluster id> cluster id\n");
  printf("-b              Bridge mode (forward in both directions)\n");
  printf("-g <core id>    Bind this app to a core (with -b use <core id>:<core id>)\n");
  printf("-a              Active packet wait\n");
  printf("-f              Flush packets immediately\n");
  printf("-v              Verbose\n");
  printf("-e              match mode\n");
  printf("-n              num of threads\n");
  printf("-N              Two NIC control\n");
  printf("-s              BlocksNumper\n");
  printf("-t              threadsperBlock\n");
  printf("-r		  buffer_times\n");
  exit(-1);
}

/* *************************************** */
//CUDA kernel function
__global__ void snort_stream_prefilter(u_char * Tx,int len,int GPUtimes,bool *Tablehits, int *whoami_GPU, u_char * fullTable, unsigned char *T1_char, unsigned char *T2_char, unsigned char *BT)
{
	int gid= blockIdx.x*blockDim.x+threadIdx.x; // 0~20,128,0~128
	int tid= threadIdx.x;
	unsigned char pak_pick1, pak_pick2, pak_pick3;
	int state=0;
	int mlist_counter=0;
	unsigned char *Tend;
	unsigned char *T;
	T=Tx+gid*len+2;
	Tend = T + len -2;
	unsigned char *whoami;
	//whoami = T - 1;
	unsigned char *CPUState;
	CPUState = T - 2;
	//printf("%d,%d,%d\n",blockIdx.x,blockDim.x,threadIdx.x);
	//printf("%d\n",blockDim.x*gridDim.x);

	int vt1_pos = 0;
	int checkBT = 0;
	int PidforT2 = 0;
	int tmp = 0;
	unsigned char *Tpre;
#if defined(_GPU_shared)	
	__shared__ unsigned char sh_T1_char[8192];
	__shared__ unsigned char sh_T2_char[6720];
	__shared__ unsigned char sh_BT[1024];
	
	if(threadIdx.x==0)
	{
		//printf("%d\n",*CPUState);
		for(int i=0;i<8192;i++)
		{
			sh_T1_char[i] = T1_char[i];
		}
		for(int i=0;i<6720;i++)
		{
			sh_T2_char[i] = T2_char[i];
		}
		for(int i=0;i<1024;i++)
		{
			sh_BT[i] = BT[i];
		}
	}

	__syncthreads();
#endif	

	if(*CPUState == 0)
	{
		for(int d_GPUtimes = 0; d_GPUtimes < GPUtimes; d_GPUtimes++)
		{
			Tpre = T + (blockDim.x*gridDim.x)*d_GPUtimes*len;
	//printf("%d, %d, %d\n",(int)*(Tpre),(int)*(Tpre+1),(int)*(Tpre+2));
		whoami = Tpre - 1;
			for (int i=0; i<1458-2; i++)
			{
#if defined(_GPU_texture)
			if( tex1Dfetch( texFulltable,(( *(Tpre+i) *256 +  *(Tpre+i+1) )*32 ) + *(Tpre+i+2) /8) & (128 >> (*(Tpre+i+2)%8 )) )
			{
				//Tablehits[gid+(blockDim.x*gridDim.x)*d_GPUtimes] = 1;
				whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes] = whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes] + ((int)*whoami+1);
				break;
			}
#elif defined(_GPU_shared)
				pak_pick1 = (int)*(Tpre+i);
				pak_pick2 = (int)*(Tpre+i+1);
				vt1_pos = ( pak_pick1 *256 ) + pak_pick2 ;
				if( d_CheckT1((int)vt1_pos,sh_T1_char))
				{
					pak_pick3 = (int)*(Tpre+i+2);
					checkBT = (int)sh_BT[vt1_pos/64];
					PidforT2 = checkBT + d_BitCount(sh_T1_char, (vt1_pos/64)*64 ,vt1_pos);
					if( d_CheckT2(PidforT2*256+(int)pak_pick3,sh_T2_char) )
					{
						//Tablehits[gid] = 1;
						//atomicAdd(&whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes],(int)*whoami+1);
						whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes] = whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes] + ((int)*whoami+1);
						//printf("%d = %d\n",gid+(blockDim.x*gridDim.x)*d_GPUtimes,(int)*whoami+1);
						break;
					}
				}
#endif				
			}
		}
	}else if(*CPUState == 1)
	{
		for(int d_GPUtimes = 0; d_GPUtimes < GPUtimes; d_GPUtimes++)
		{
			//whoami = T + (blockDim.x*gridDim.x)*d_GPUtimes*len -1;
			//whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes] = whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes] + ((int)*whoami+1);
			//atomicAdd(&whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes],(int)*whoami+1);
		}
	}
}
__global__ void snort_stream_snort(u_char * Tx, ACSM_STRUCT * acsm,int len,int GPUtimes,int *whoami_GPU, int *nfound_GPU, u_char * fullTable)
{
	int gid= blockIdx.x*blockDim.x+threadIdx.x; // 20,128,0~128
	//printf("%d, %d\n",blockIdx.x,threadIdx.x);
	int tid= threadIdx.x;
	unsigned char pak_pick1, pak_pick2, pak_pick3;
	int state=0;
	int mlist_counter=0;
	__shared__	ACSM_PATTERN * mlist;
	unsigned char *Tend;
	__shared__	ACSM_STATETABLE * StateTable;
	StateTable= acsm->acsmStateTable;
	unsigned char *T;
	T=Tx+gid*len+2;
	Tend = T + len -2;
	unsigned char *Tsnort;
	unsigned char *whoami;
	
	
	bool SkipPre = false;
	for(int d_GPUtimes = 0;d_GPUtimes<GPUtimes;d_GPUtimes++)
	{
		Tsnort = T + (blockDim.x*gridDim.x)*d_GPUtimes*len;
		whoami = Tsnort -1;
	//printf("%d, %d\n",(int)*Tsnort, (int)*(Tsnort+1));
	//printf("%d\n",(int)*(Tsnort-1));
		for (int i=0;i<1458;i++)
		{	
			// into T2 times counting
			state = StateTable[state].NextState[*(Tsnort+i)];

			if( StateTable[state].MatchList != NULL )
			{
				//printf("%d,%d\n",i,StateTable[state].MatchList->n);
				//whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes] = whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes] + ((int)*whoami+1);
				for( mlist=StateTable[state].MatchList; mlist!=NULL;mlist=mlist->next )
				{
					//nfound_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes]=nfound_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes]+1;
					whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes] = whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes] + ((int)*whoami+1);
					//atomicAdd(&nfound_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes],1);
				}
			}
		}
	}
	//atomicAdd(&whoami_GPU[gid],(int)*whoami+1);

}
//GGG
__global__ void snort_ac_prefilter(u_char * Tx, ACSM_STRUCT * acsm,int len, int GPUtimes, int *nfound_GPU,bool *Tablehits, int *whoami_GPU, u_char * fullTable, unsigned char *T1_char, unsigned char *T2_char, unsigned char *BT)
{
	int gid= blockIdx.x*blockDim.x+threadIdx.x; // 20,128,0~128
	int tid= threadIdx.x;
	unsigned char pak_pick1, pak_pick2, pak_pick3;
	int state=0;
	int mlist_counter=0;
	__shared__	ACSM_PATTERN * mlist;
	unsigned char *Tend;
	__shared__	ACSM_STATETABLE * StateTable;
	StateTable= acsm->acsmStateTable;
	unsigned char *T;
	T=Tx+gid*len+2;
	Tend = T + len -2;
	unsigned char *whoami;
	whoami = T - 1;
	unsigned char *CPUState;
	CPUState = T - 2;
	//printf("%d, %d, %d\n",blockIdx.x,blockDim.x,threadIdx.x); 20,128,0~128
	//printf("%d\n",gid);
	//printf("%d: %d, %d, %d\n", gid,*T,*(T+1),*(T+2));  //work
	//printf("%d\n",(int)Tend-(int)T);

	int vt1_pos = 0;
	int checkBT = 0;
	int PidforT2 = 0;
	int tmp = 0;
	unsigned char *Tpre;
	unsigned char *Tsnort;
	Tpre = T;
#if defined(_GPU_shared)
	__shared__ unsigned char sh_T1_char[8192];
	__shared__ unsigned char sh_T2_char[6720];
	__shared__ unsigned char sh_BT[1024];
	
	if(threadIdx.x==0)
	{
		//printf("%d\n",*CPUState);
		for(int i=0;i<8192;i++)
		{
			sh_T1_char[i] = T1_char[i];
		}
		for(int i=0;i<6720;i++)
		{
			sh_T2_char[i] = T2_char[i];
		}
		for(int i=0;i<1024;i++)
		{
			sh_BT[i] = BT[i];
		}
	}

	__syncthreads();
#endif	
	
	bool SkipPre = false;
	//for (state = 0; T < Tend ; T++)
	for(int d_GPUtimes = 0;d_GPUtimes<GPUtimes;d_GPUtimes++)
	{
		Tsnort = T + (blockDim.x*gridDim.x)*d_GPUtimes*len;
		for (int i=0;i<1458;i++)
		{	
			// into T2 times counting
			//state = StateTable[state].NextState[*T];
			state = StateTable[state].NextState[*(Tsnort+i)];
			
			if( StateTable[state].MatchList != NULL )
			{
				for( mlist=StateTable[state].MatchList; mlist!=NULL;mlist=mlist->next )
				{
					//nfound_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes]=nfound_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes]+1;
					whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes] = whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes] + ((int)*whoami+1);
					//nfound_GPU[gid]=nfound_GPU[gid]+1;
					//nfound_GPU[gid] = mlist->iid;
					////char		nfound_GPU[id*4+mlist_counter]= mlist->iid;
				}
			}
	
		}
	}
	
	if(*CPUState == 0)
	{
		for(int d_GPUtimes = 0; d_GPUtimes < GPUtimes; d_GPUtimes++)
		{
			Tpre = T + (blockDim.x*gridDim.x)*d_GPUtimes*len;
			whoami = Tpre -1;
			for (int i=0; i<1458-2; i++)
			{
#if defined(_GPU_shared)				
				pak_pick1 = (int)*(Tpre+i);
				pak_pick2 = (int)*(Tpre+i+1);
				//vt1_pos = ( *(Tpre+i) << 8 ) | *(Tpre+i+1) ;
				vt1_pos = ( pak_pick1 *256 ) + pak_pick2 ;
				if( d_CheckT1((int)vt1_pos,sh_T1_char))
				{
					//printf("%d, %d, %d\n",(int)*whoami,(int)pak_pick1,(int)pak_pick2);
					pak_pick3 = (int)*(Tpre+i+2);
					//Tablehits[gid]=1;
					//nfound_GPU[gid] = nfound_GPU[gid]+1;
					checkBT = (int)sh_BT[vt1_pos/64];
					PidforT2 = checkBT + d_BitCount(sh_T1_char, (vt1_pos/64)*64 ,vt1_pos);
					if( d_CheckT2(PidforT2*256+(int)pak_pick3,sh_T2_char) )
					{
						//printf("%d, %d, %d\n",(int)pak_pick1,(int)pak_pick2,(int)pak_pick3);
						//printf("whoami: %d\n",(int)*whoami);
		
						//Tablehits[gid] = 1;
						//nfound_GPU[gid] = nfound_GPU[gid]+1;
						//whoami_GPU[gid]=(int)*whoami+1;
					
						//atomicAdd(&whoami_GPU[gid],(int)*whoami+1);
						whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes] = whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes] + ((int)*whoami+1);
		
						//__syncthreads();
						break;
					}
				}
#elif defined(_GPU_texture)
				if( tex1Dfetch( texFulltable,(( *(Tpre+i) *256 +  *(Tpre+i+1) )*32 ) + *(Tpre+i+2) /8) & (128 >> (*(Tpre+i+2)%8 )) )
				{
					//Tablehits[gid+(blockDim.x*gridDim.x)*d_GPUtimes] = 1;
					whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes] = whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes] + ((int)*whoami+1);
					break;
				}
#endif				
			}
		}
	}else if(*CPUState == 1)
	{
		for(int d_GPUtimes = 0; d_GPUtimes < GPUtimes; d_GPUtimes++)
		{
			whoami = T + (blockDim.x*gridDim.x)*d_GPUtimes*len -1;
			//atomicAdd(&whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes],(int)*whoami+1);
			whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes] = whoami_GPU[gid+(blockDim.x*gridDim.x)*d_GPUtimes] + ((int)*whoami+1);
		}
	}





}

//CPU presistent PPP
void *persistentkernel(void *unused)
{
	hipStream_t stream1, stream2, stream3, stream4;
	hipEvent_t event,start1,start2,stop1,stop2;

	/*hipEventCreate(&event);
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	hipEventCreate(&start2);
	hipEventCreate(&stop2);
	float time1=0, time2=0;*/

	int GPUtimes = buffer_size/(blocksNumper*threadsperBlock);
	cout << "GPUtimes: " << GPUtimes << endl;
	
	bind2core(7);
	long long int totalcount = 0;
	while(!do_shutdown){
		usleep(1);
		if(!isReady[0])
		{
			/*hipStreamCreate(&stream1);
			hipStreamCreate(&stream2);
			hipStreamCreate(&stream3);
			hipStreamCreate(&stream4);*/

			totalcount=0;
			gettimeofday(&startCPU,NULL);
			for(int i=0;i<num_threads;i++)
			{
				//memcpy(Central_memory+totalcount,CPU_Central_memory+(i*bytes),CPU_Count[i]*buffershift);
				fastMemcpy(Central_memory+totalcount,CPU_Central_memory+(i*bytes),CPU_Count[i]*buffershift);
				totalcount+=CPU_Count[i]*buffershift;
			}
			totalcount = 0;
			
			for(int i=0;i<num_threads;i++)
			{
				CPU_Count[i] = 0;
			}
			isReady[0] = true;
			cv_buffer1.notify_all();
			
			gettimeofday(&endCPU,NULL);
			diffCPU += 1000000*(endCPU.tv_sec-startCPU.tv_sec)+endCPU.tv_usec-startCPU.tv_usec;
			//cout << "                                 time1: " << diffCPU << endl;
			diffCPU = 0;

			

			hipMemcpy(d_Central_memory,Central_memory,bytes,hipMemcpyHostToDevice);
			//cv_buffer1.notify_all();
			
			/*hipMemcpy(d_T1_char,&T1_char,65536/8,hipMemcpyHostToDevice);
			hipMemcpy(d_T2_char,&T2_char,53760/8,hipMemcpyHostToDevice);
			hipMemcpy(d_BT_char,BT,1024,hipMemcpyHostToDevice);
			snort_ac_prefilter<<<blocksNumper,threadsperBlock>>>(d_Central_memory,acsm,(payloadlen+3),d_nfound_GPU1,d_Tablehits1,d_whoami_GPU,d_fullTable,d_T1_char,d_T2_char,d_BT_char);
			hipMemcpy(Tablehits1, d_Tablehits1,sizeof(bool)*(blocksNumper*threadsperBlock), hipMemcpyDeviceToHost);
			hipMemcpy(whoami_GPU, d_whoami_GPU,sizeof(int)*(blocksNumper*threadsperBlock), hipMemcpyDeviceToHost);
			*/
	#if defined(_GPU_shared)		
			/*hipMemcpyAsync(d_T1_char,&T1_char,65536/8,hipMemcpyHostToDevice);
			hipMemcpyAsync(d_T2_char,&T2_char,53760/8,hipMemcpyHostToDevice);
			hipMemcpyAsync(d_BT_char,BT,1024,hipMemcpyHostToDevice);*/
	#endif			
			//snort_stream_prefilter<<<blocksNumper,threadsperBlock,0>>>(d_Central_memory,buffershift,GPUtimes,d_Tablehits1,d_whoami_GPU,d_fullTable,d_T1_char,d_T2_char,d_BT_char);
			snort_stream_snort<<<blocksNumper,threadsperBlock,0>>>(d_Central_memory,acsm,buffershift,GPUtimes,d_whoami_GPU,d_nfound_GPU1,d_fullTable);
			//snort_ac_prefilter<<<blocksNumper,threadsperBlock,0>>>(d_Central_memory,acsm,buffershift,GPUtimes,d_nfound_GPU1,d_Tablehits1,d_whoami_GPU,d_fullTable,d_T1_char,d_T2_char,d_BT_char);
			
			//snort_stream_prefilter<<<blocksNumper,threadsperBlock,0,stream1>>>(d_Central_memory,buffershift,GPUtimes,d_Tablehits1,d_whoami_GPU,d_fullTable,d_T1_char,d_T2_char,d_BT_char);
			//hipEventRecord(event,stream1);
			//snort_stream_snort<<<blocksNumper,threadsperBlock,0,stream2>>>(d_Central_memory,acsm,buffershift,GPUtimes,d_nfound_GPU1,d_fullTable);
			
			//hipStreamWaitEvent(stream3,event,0);
			//hipMemcpyAsync(whoami_GPU, d_whoami_GPU,sizeof(int)*(blocksNumper*threadsperBlock*buffer_times), hipMemcpyDeviceToHost,stream3);

			//hipMemcpy(Tablehits1, d_Tablehits1,sizeof(bool)*(blocksNumper*threadsperBlock*buffer_times), hipMemcpyDeviceToHost);
			
			hipMemcpy(whoami_GPU, d_whoami_GPU,sizeof(int)*(blocksNumper*threadsperBlock*buffer_times), hipMemcpyDeviceToHost);
			//hipMemcpy(nfound_GPU1, d_nfound_GPU1,sizeof(int)*(blocksNumper*threadsperBlock*buffer_times), hipMemcpyDeviceToHost);

			hipDeviceSynchronize();
			/*hipEventSynchronize(stop1);
			hipEventElapsedTime(&time1, start1, stop1);
			outputyoGPU << time1 << endl;*/

			/*hipStreamDestroy(stream1);
			hipStreamDestroy(stream2);
			hipStreamDestroy(stream3);
			hipStreamDestroy(stream4);*/

			intoGPUtime++;
			countGPUthreadtimes++;
			
			for(int i=0;i<blocksNumper*threadsperBlock*buffer_times;i++)
			{
				countpak[i%num_threads]++;
				switch (whoami_GPU[i])
				{
					case 0:
						break;
					case 1:
						count_intermittent[0]++;
						temp_test[0]++;
						break;
					case 2:
						count_intermittent[1]++;
						temp_test[0]++;
					        break;
					case 3:
						count_intermittent[2]++;
						temp_test[0]++;
					        break;
					case 4:
						count_intermittent[3]++;
						temp_test[0]++;
					        break;
					case 5:
						count_intermittent[4]++;
						temp_test[0]++;
					        break;
					case 6:
						count_intermittent[5]++;
						temp_test[0]++;
					        break;
					case 7:
						count_intermittent[6]++;
						temp_test[0]++;
						break;
				}
			}
			// count percentage to call CPU

			/*for(int i=0;i<blocksNumper*threadsperBlock*buffer_times;i++)
			{
				//cout << nfound_GPU1[i] << endl;
				//temp_test[0]=temp_test[0]+(nfound_GPU1[i]);
				//temp_test[2]=temp_test[2]+(Tablehits1[i]);
			}*/
			kernel_sum+=temp_test[0];
			//kernel_tablehits+=temp_test[2];
			//cout << "kernel_sum1: " << temp_test[0] << endl;
			CountNumofPacketToGpuFun1++;
			temp_test[0]=0;
			temp_test[2]=0;

			//hipMemset(d_nfound_GPU1,0,sizeof(int)*blocksNumper*threadsperBlock*buffer_times);
			//hipMemset(d_Tablehits1,0,sizeof(bool)*blocksNumper*threadsperBlock*buffer_times);
			hipMemset(d_whoami_GPU,0,sizeof(int)*blocksNumper*threadsperBlock*buffer_times);
			memset(nfound_GPU1,0,sizeof(int)*blocksNumper*threadsperBlock*buffer_times);

		}else if(!isReady[1])
		{
			/*hipStreamCreate(&stream1);
			hipStreamCreate(&stream2);
			hipStreamCreate(&stream3);
			hipStreamCreate(&stream4);*/

			totalcount=0;
			gettimeofday(&startCPU,NULL);
			for(int i=0;i<num_threads;i++)
			{
				//memcpy(Central_memory+bytes+totalcount,CPU_Central_memory+((i+num_threads)*bytes),CPU_Count[i+num_threads]*buffershift);
				fastMemcpy(Central_memory+bytes+totalcount,CPU_Central_memory+((i+num_threads)*bytes),CPU_Count[i+num_threads]*buffershift);
				totalcount+=CPU_Count[i+num_threads]*buffershift;
				//cout << CPU_Count[i+num_threads] << " ";
			}
			totalcount = 0;

			for(int i=0;i<num_threads;i++)
			{
				CPU_Count[i+num_threads] = 0;
			}
			isReady[1] = true;
			cv_buffer2.notify_all();
			
			//cout << endl;
			gettimeofday(&endCPU,NULL);
			diffCPU += 1000000*(endCPU.tv_sec-startCPU.tv_sec)+endCPU.tv_usec-startCPU.tv_usec;
			//cout << "                                 time2: " << diffCPU << endl;
			diffCPU = 0;

			

			hipMemcpy(d_Central_memory+bytes,Central_memory+bytes,bytes,hipMemcpyHostToDevice);
			//cv_buffer2.notify_all();

			/*hipMemcpy(d_T1_char,&T1_char,65536/8,hipMemcpyHostToDevice);
			hipMemcpy(d_T2_char,&T2_char,53760/8,hipMemcpyHostToDevice);
			hipMemcpy(d_BT_char,BT,1024,hipMemcpyHostToDevice);
			snort_ac_prefilter<<<blocksNumper,threadsperBlock>>>(d_Central_memory+bytes,acsm,(payloadlen+3),d_nfound_GPU2,d_Tablehits2,d_whoami_GPU,d_fullTable,d_T1_char,d_T2_char,d_BT_char);
			hipMemcpy(Tablehits2, d_Tablehits2,sizeof(bool)*(blocksNumper*threadsperBlock), hipMemcpyDeviceToHost);
			hipMemcpy(whoami_GPU, d_whoami_GPU,sizeof(int)*(blocksNumper*threadsperBlock), hipMemcpyDeviceToHost);
			*/
	#if defined(_GPU_shared)
			/*hipMemcpyAsync(d_T1_char,&T1_char,65536/8,hipMemcpyHostToDevice);
			hipMemcpyAsync(d_T2_char,&T2_char,53760/8,hipMemcpyHostToDevice);
			hipMemcpyAsync(d_BT_char,BT,1024,hipMemcpyHostToDevice);*/
	#endif

			//snort_stream_prefilter<<<blocksNumper,threadsperBlock,0>>>(d_Central_memory+bytes,buffershift,GPUtimes,d_Tablehits2,d_whoami_GPU,d_fullTable,d_T1_char,d_T2_char,d_BT_char);
			snort_stream_snort<<<blocksNumper,threadsperBlock,0>>>(d_Central_memory+bytes,acsm,buffershift,GPUtimes,d_whoami_GPU,d_nfound_GPU2,d_fullTable);	
			//snort_ac_prefilter<<<blocksNumper,threadsperBlock,0>>>(d_Central_memory,acsm,buffershift,GPUtimes,d_nfound_GPU2,d_Tablehits2,d_whoami_GPU,d_fullTable,d_T1_char,d_T2_char,d_BT_char);
				
			//snort_stream_prefilter<<<blocksNumper,threadsperBlock,0,stream1>>>(d_Central_memory+bytes,buffershift,GPUtimes,d_Tablehits2,d_whoami_GPU,d_fullTable,d_T1_char,d_T2_char,d_BT_char);
			//hipEventRecord(event,stream1);
			//snort_stream_snort<<<blocksNumper,threadsperBlock,0,stream2>>>(d_Central_memory+bytes,acsm,buffershift,GPUtimes,d_nfound_GPU2,d_fullTable);
			//hipStreamWaitEvent(stream3,event,0);
			//hipMemcpyAsync(whoami_GPU, d_whoami_GPU,sizeof(int)*(blocksNumper*threadsperBlock*buffer_times), hipMemcpyDeviceToHost,stream3);
				
			//hipMemcpy(Tablehits2, d_Tablehits2,sizeof(bool)*(blocksNumper*threadsperBlock*buffer_times), hipMemcpyDeviceToHost);

			hipMemcpy(whoami_GPU, d_whoami_GPU,sizeof(int)*(blocksNumper*threadsperBlock*buffer_times), hipMemcpyDeviceToHost);
			//hipMemcpy(nfound_GPU2,d_nfound_GPU2,sizeof(int)*(blocksNumper*threadsperBlock*buffer_times), hipMemcpyDeviceToHost);

			hipDeviceSynchronize();
			/*hipEventSynchronize(stop2);
			hipEventElapsedTime(&time2, start2, stop2);
			outputyoGPU << time2 << endl;*/

			/*hipStreamDestroy(stream1);
			hipStreamDestroy(stream2);
			hipStreamDestroy(stream3);
			hipStreamDestroy(stream4);*/

			intoGPUtime++;
			countGPUthreadtimes++;
			
			for(int i=0;i<blocksNumper*threadsperBlock*buffer_times;i++)
			{
				countpak[i%num_threads]++;
			        switch (whoami_GPU[i])
			        {
					case 0: 
						break;
			                case 1:
						count_intermittent[0]++;
						temp_test[1]++;
			                        break;
			                case 2:
						count_intermittent[1]++;
						temp_test[1]++;
			                        break;
			                case 3:
						count_intermittent[2]++;
						temp_test[1]++;
			                        break;
			                case 4:
						count_intermittent[3]++;
						temp_test[1]++;
			                        break;
			                case 5:
						count_intermittent[4]++;
						temp_test[1]++;
			                        break;
			                case 6:
						count_intermittent[5]++;
						temp_test[1]++;
			                        break;
					case 7:
						count_intermittent[6]++;
						temp_test[1]++;
						break;
			        }
			}
			
			// count percentage to call CPU
			//int threadpak = intoGPUtime*blocksNumper*threadsperBlock/num_threads;
			/*if(intoGPUtime>=numofpakcal)
			{
				cout < "fu3k you" << endl;
				for(int i=0;i<num_threads;i++)
				{
					/*if( ((double)temp_whoami[i] / threadpak) > AdaptiveThreashold )  //AdaptiveThreashold = 0.8;
					{
						//cout << "1CPU" << i << "  is time to close prefilter! cuz now percent is:" << (double)temp_whoami[i] / threadpak << endl;
						if(CPUPrefilter[i] == false)
						{
							//CPUPrefilter[i]=false; // pre -> nonpre
							//cout << "GPU: 1close!"<< (double)temp_whoami[i] << "/" << threadpak << "= " << (double)temp_whoami[i] / threadpak << endl;
						}
					}else if( ((double)temp_whoami[i] / threadpak) < AdaptiveThreashold )  //AdaptiveThreashold = .8;
					{
						if(CPUPrefilter[i] == false)
						{
							//CPUPrefilter[i]=true;
							//cout << "GPU: 1open!"<< (double)temp_whoami[i] << "/" << threadpak << "= " << (double)temp_whoami[i] / threadpak << endl;
						}
					}
					temp_whoami[i] = 0;*/
					/*if(CPUPrefilter[i] == false)
					{
						CPUPrefilter[i] = true;
					}
				}
				intoGPUtime=0;
			}*/

			/*for(int i=0;i<blocksNumper*threadsperBlock*buffer_times;i++)
			{
				//temp_test[1]=temp_test[1]+(nfound_GPU2[i]);
				//temp_test[3]=temp_test[3]+(Tablehits2[i]);
			}*/
			kernel_sum += temp_test[1];
			//kernel_tablehits += temp_test[3];
			//cout << "kernel_sum2: " << temp_test[1] << endl;
			CountNumofPacketToGpuFun2++;
			temp_test[1]=0;
			temp_test[3]=0;

			//hipMemset(d_nfound_GPU2,0,sizeof(int)*blocksNumper*threadsperBlock*buffer_times);
			//hipMemset(d_Tablehits2,0,sizeof(bool)*blocksNumper*threadsperBlock*buffer_times);
			hipMemset(d_whoami_GPU,0,sizeof(int)*blocksNumper*threadsperBlock*buffer_times);
			memset(nfound_GPU2,0,sizeof(int)*blocksNumper*threadsperBlock*buffer_times);
			
		}
	
	}


	pthread_exit(NULL);
}

//CPU function
void *packet_consumer_thread(void *_i) {
	struct dir_info *i = (struct dir_info *) _i;
	int tx_queue_not_empty = 0;
	u_char* buffer;
	//gpu
	pthread_t thread4gpu[8];
	//time
	struct timeval start;
	struct timeval end;
	unsigned long diff = 0;
	//acsm
	int nfound=0;

	int check = 0;
	int pos;
	unsigned char idx;
	unsigned char each_T1_char[sizeof(T1_char)];
	unsigned char each_T2_char[sizeof(T2_char)];
	unsigned char each_BT[btsize*sizeof(char)];
	
	memcpy(each_T1_char,T1_char,sizeof(T1_char));
	memcpy(each_T2_char,T2_char,sizeof(T2_char));
	memcpy(each_BT,BT,btsize*sizeof(char));

	if (i->bind_core >= 0)
	{
		bind2core(i->bind_core);
	}
  ////////////////////////////////////////
#if defined(_hpma)
	cout << "In _hpma" << endl;
#elif defined(_GPU)
	cout << "In _GPU" << endl;
//#elif defined(_CPU)
	//cout << "In _CPU" << endl;	
#endif
		int whoami = i->bind_core;
		//cout << "whoami: " << whoami << endl;
		int counta=0, countf=0, counts=0;
		int counterbuffer=0;
		int checkbyT1 = 0;
		u_char* temp;
		u_char* __restrict__ CPU_Central_memory_tail; 
		CPU_Central_memory_tail = CPU_Central_memory + (whoami*bytes);
		int countpat1=0,counterT1=0,counter3=0;
		unsigned char pak_pick1, pak_pick2,pak_pick3;
		int vt1_pos=0,count_t1_locat=0, win_no=0;
		int temp_test=0;
		int PidforT2=0;
		int checkBT=0;
		double AP = 0.0;
		bool CPUState = 1;
		while(!do_shutdown) 
		{
			if(pfring_zc_recv_pkt(i->inzq, &i->tmpbuff, wait_for_packet) > 0) 
			{

				if (unlikely(verbose)) 
				{
					char strbuf[4096];
					int strlen = sizeof(strbuf);
					int strused = snprintf(strbuf, strlen, "[%s -> %s]", i->in_dev, i->out_dev);
					pfring_print_pkt(&strbuf[strused], strlen - strused, pfring_zc_pkt_buff_data(i->tmpbuff, i->inzq), i->tmpbuff->len, i->tmpbuff->len);
					fputs(strbuf, stdout);
				}
				buffer = pfring_zc_pkt_buff_data(i->tmpbuff, i->inzq)+42;
				i->numPkts++;
				i->numBytes += i->tmpbuff->len + 24; /* 8 Preamble + 4 CRC + 12 IFG */
				  
				errno = 0;
				while (unlikely(pfring_zc_send_pkt(i->outzq, &i->tmpbuff, flush_packet) < 0 && errno != EMSGSIZE && !do_shutdown)) //send packet
				if (wait_for_packet)
				{					
					usleep(1);
				}
				
				tx_queue_not_empty = 1;
			}else {
				if (tx_queue_not_empty) 
				{
					pfring_zc_sync_queue(i->outzq, tx_only);
					tx_queue_not_empty = 0;
				}
				if (wait_for_packet) 
				{
					usleep(1);
				}
			}
			gettimeofday(&start,NULL); //666
//#if defined(_CPU) // pure CPU acsm
			//nfound+=acsmSearch(acsm, buffer, payloadlen, MatchFound, 0); //pure CPU

			// hpma prefilter start 
			//unique_lock<mutex> lk(lockurmother,defer_lock);
			bool checkifT1 = false;
			bool inT2 = false;
			if( CPUPrefilter[whoami] == true )
			{
				CPUState = 1;
				//unique_lock<mutex> lk(lockurmother);
				totalpacket[whoami]++;
				//lk.unlock();
				for(int i = 0 ; i < payloadlen-2 ; i=i+1 )
				{
					pak_pick1 = *(buffer+i);
					pak_pick2 = *(buffer+i+1);
					vt1_pos = (pak_pick1<<8) |pak_pick2;  //pak_pick1*256+pak_pick2
					if( CheckT1((int)vt1_pos) ) // Prefilter T1
					/*pos = (int)vt1_pos;
					idx = each_T1_char[pos >> 3]; // pos/8
					check = pos & 0x07;
					if( idx & ( 0x01 << (7-check) ) )*/
					{
						checkifT1 = true;
						counterT1++;
						pak_pick3 = *(buffer+i+2);
						
						checkBT = (int)each_BT[vt1_pos/64];
						//checkBT = (int)BT[vt1_pos/64];
						PidforT2 = checkBT + BitCount(each_T1_char, (vt1_pos & 0xfffffc0) ,vt1_pos);
						//PidforT2 = checkBT + BitCount(T1_char, (vt1_pos & 0xfffffc0) ,vt1_pos);
						
						if( CheckT2(PidforT2*256+(int)pak_pick3) )  // Prefilter T2
						/*pos = (PidforT2*256+(int)pak_pick3);
						idx = each_T2_char[pos >> 3]; // pos/8
						check = pos & 0x07;
						if( idx & (0x01 << (7-check) ) )*/
						{
							//CPUbyte[whoami]+=i;
							inT2 = true;
							counterT2[whoami]++;
					
							//no each buffer
							/*if(DB_Check == 1)
							{
								if(!isReady[1])
								{
									unique_lock<mutex> lk(lockurmother);
									while(!isReady[1])
									{
										cv_buffer2.wait(lk);
									}
									lk.unlock();
								}
									
								sem_wait(&os_sem);

								temp = Central_memory_tail;
								
								memcpy(temp,&CPUState,1);
								memcpy(temp+1,&whoami,1);
								memcpy(temp+2,buffer,payloadlen);

								Central_memory_tail = Central_memory_tail + buffershift;
								
								if(Central_memory_tail == second_buffer)
								{
									isReady[1] == false;
									Central_memory_tail = Central_memory;
									DB_Check = !DB_Check;
								}
								sem_post(&os_sem);

							}else if(DB_Check == 0)
							{
								if(!isReady[0])
								{
									unique_lock<mutex> lk(lockurmother);
									while(!isReady[0])
									{
										cv_buffer1.wait(lk);
									}
									lk.unlock();
								}

								sem_wait(&os_sem);

								temp = Central_memory_tail;
								
								memcpy(temp,&CPUState,1);
								memcpy(temp+1,&whoami,1);
								memcpy(temp+2,buffer,payloadlen);

								Central_memory_tail = Central_memory_tail + buffershift;
								
								if(Central_memory_tail == first_buffer)
								{
									isReady[0] == false;
									DB_Check = !DB_Check;
								}
								sem_post(&os_sem);

							}*/
															//


							if(isReady[0] == isReady[1] && isReady[1] == 0)
							{
								alldead++;
							}

							if(DB_Check == 1)
							{
								if(!isReady[1])
								{
									unique_lock<mutex> lk(lockurmother);
									while(!isReady[1])
									{
										cv_buffer2.wait(lk);
									}
									lk.unlock();
								}
								
								//temp = CPU_Central_memory_tail + (num_threads*bytes) + (CPU_Count[whoami+num_threads]*1472);
								temp = Buffer_location[CPU_Count[whoami+num_threads]];
								temp = temp + (whoami*bytes) + (num_threads*bytes);

								memcpy(temp,&CPUState,1);
								memcpy(temp+1,&whoami,1);
								memcpy(temp+2,buffer,payloadlen);

								CPU_Count[whoami+num_threads]++;

							}else if(DB_Check == 0)
							{
								if(!isReady[0])
								{
									unique_lock<mutex> lk(lockurmother);
									while(!isReady[0])
									{
										cv_buffer1.wait(lk);
									}
									lk.unlock();
									
								}
				
								//temp = CPU_Central_memory_tail + (CPU_Count[whoami]*1472);
								temp = Buffer_location[CPU_Count[whoami]];
								temp = temp + whoami*bytes;
			
								memcpy(temp,&CPUState,1);
								memcpy(temp+1,&whoami,1);
								memcpy(temp+2,buffer,payloadlen);
									
								CPU_Count[whoami]++;

							}
							
							unique_lock<mutex> lk(lockurmother);
							//unique_lock<mutex> lk(lockurmother,defer_lock);
						//if(lk.try_lock())
						//{
							DB_Count++;
							if(DB_Count >= buffer_size)
							{
								if(DB_Check==1)
								{
									isReady[1] = false;
									for(int i=0;i<num_threads;i++)
									{
										//CPU_Count[i] = 0;
									}
									
								}else if(DB_Check==0)
								{
									isReady[0] = false;
									for(int i=0;i<num_threads;i++)
									{
										//CPU_Count[i+num_threads] = 0;
									}
								}
								DB_Check = !DB_Check;
								DB_Count = 0;
							}
							justanumber[whoami]++;
							//totallock[whoami]++;
						/*}else{
							totalcantlock[whoami]++;
							lk.lock();
							DB_Count++;
							if(DB_Count >= buffer_size)
							{
								if(DB_Check==1)
								{
									isReady[1] = false;
									for(int i=0;i<num_threads;i++)
									{
										//CPU_Count[i] = 0;
									}
									
								}else if(DB_Check==0)
								{
									isReady[0] = false;
									for(int i=0;i<num_threads;i++)
									{
										//CPU_Count[i+num_threads] = 0;
									}
								}
								DB_Check = !DB_Check;
								DB_Count = 0;
							}
						}*/
							lk.unlock();
							
							break;
						}
					}
				}
				if(inT2 == false)
				{
					countpak[whoami]++;
					//CPUbyte[whoami]+=1458;
				}else if (inT2 == true)
				{
				
				}
			}
			else if ( CPUPrefilter[whoami] == false )
			{
				if(isReady[0] == isReady[1] && isReady[1] == 0)
				{
					alldead++;
				}
				CPUState = 0;
				if(DB_Check == 1)
				{
					if(!isReady[1])
					{
						unique_lock<mutex> lk(lockurmother);
						while(!isReady[1])
						{
							cv_buffer2.wait(lk);
						}
						lk.unlock();
					}

					//temp = CPU_Central_memory_tail + (num_threads*bytes) + (CPU_Count[whoami+num_threads]*1461);
					temp = Buffer_location[CPU_Count[whoami+num_threads]];
					temp = temp + (whoami*bytes) + (num_threads*bytes);

					memcpy(temp,&CPUState,1);
					memcpy(temp+1,&whoami,1);
					memcpy(temp+2,buffer,payloadlen);
						
					CPU_Count[whoami+num_threads]++;
					
				}else if(DB_Check == 0)
				{
					if(!isReady[0])
					{
						unique_lock<mutex> lk(lockurmother);
						while(!isReady[0])
						{
							cv_buffer1.wait(lk);
						}
						lk.unlock();
					}
					
					//temp = CPU_Central_memory_tail + (CPU_Count[whoami]*1461);
					temp = Buffer_location[CPU_Count[whoami]];
					temp = temp + whoami*bytes;

					memcpy(temp,&CPUState,1);
					memcpy(temp+1,&whoami,1);
					memcpy(temp+2,buffer,payloadlen);
						
					CPU_Count[whoami]++;
				}
				
				///
				unique_lock<mutex> lk(lockurmother);
				//unique_lock<mutex> lk(lockurmother,defer_lock);
			//if(lk.try_lock())
			//{
				DB_Count++;
				if(DB_Count >= buffer_size)
				{
					if(DB_Check==1)
					{
						isReady[1] = false;
						for(int i=0;i<num_threads;i++)
						{
							//CPU_Count[i] = 0;
						}
					}else if(DB_Check==0)
					{
						isReady[0] = false;
						for(int i=0;i<num_threads;i++)
						{
							//CPU_Count[i+num_threads] = 0;
						}
					}
					DB_Check = !DB_Check;
					DB_Count = 0;
				}
				justanumber[whoami]++;
				//totallock[whoami]++;
			/*}else{
				
				totalcantlock[whoami]++;
				lk.lock();
				DB_Count++;
				if(DB_Count >= buffer_size)
				{
					if(DB_Check==1)
					{
						isReady[1] = false;
						for(int i=0;i<num_threads;i++)
						{
							//CPU_Count[i] = 0;
						}
					}else if(DB_Check==0)
					{
						isReady[0] = false;
						for(int i=0;i<num_threads;i++)
						{
							//CPU_Count[i+num_threads] = 0;
						}
					}
					DB_Check = !DB_Check;
					DB_Count = 0;
				}
			}*/
				lk.unlock();
				
				//justanumber[whoami]++; //debug
			}

			if(checkifT1 == true)
			{
				checkbyT1++;
			}
			/*sem_wait(&os_sem);
			pthread_join(thread4gpu[0], NULL);
			pthread_join(thread4gpu[1], NULL);
			sem_post(&os_sem);*/
			
			gettimeofday(&end,NULL);
			diff += 1000000*(end.tv_sec-start.tv_sec)+end.tv_usec-start.tv_usec;
		}
		
		if (!flush_packet) //after exit
		{
			pfring_zc_sync_queue(i->outzq, tx_only);
		}
		pfring_zc_sync_queue(i->inzq, rx_only);


		int CNOPTG = CountNumofPacketToGpuFun1+CountNumofPacketToGpuFun2 ;
		//sem_wait(&os_sem);
		printf("totalpacket= %d, countT1= %d, checkbyT1= %d, countT2= %d, countGPU= %d, GPUtablehits= %d, Pak2GPU= %d+%d=%d, pre_time= %d, GPUpre_time= %d, cudathread_time= %d\n",totalpacket, counterT1,checkbyT1,counterT2,kernel_sum, kernel_tablehits, CountNumofPacketToGpuFun1, CountNumofPacketToGpuFun2,intoGPUtime, diff, diffGPU, diffCUDA);
		//sem_post(&os_sem);
  
  return NULL;
}

/* *************************************** */

int init_direction(struct dir_info *i, char *in_dev, char *out_dev) {
  
  cout<<" in: "<<in_dev<<" out: "<<out_dev<<endl;

  i->in_dev = in_dev;
  i->out_dev = out_dev;

  i->tmpbuff = pfring_zc_get_packet_handle(zc);

  if (i->tmpbuff == NULL) {
    fprintf(stderr, "pfring_zc_get_packet_handle error\n");
    return -1;
  }

  i->inzq = pfring_zc_open_device(zc, in_dev, rx_only, 0);

  if(i->inzq == NULL) {
    fprintf(stderr, "pfring_zc_open_device error [%s] Please check that %s is up and not already used\n",
     	    strerror(errno), in_dev);
    return -1;
  }

  i->outzq = pfring_zc_open_device(zc, out_dev, tx_only, 0);

  if(i->outzq == NULL) {
    fprintf(stderr, "pfring_zc_open_device error [%s] Please check that %s is up and not already used\n",
	    strerror(errno), out_dev);
    return -1;
  }

  return 0;
}

/* *************************************** */

int main(int argc, char* argv[]) {
	pthread_t persistentyo;
  char *device1 = NULL, *device2 = NULL;
  char *device1_rss[num_threads], *device2_rss[num_threads];
  char *device1_rss_NIC[num_threads], *device2_rss_NIC[num_threads];
  char *bind_mask = NULL, c;
  long i, j, k, m, n;
  stringstream ss;
  string idder;
  char *device_at     = "@";
  char *device_10_in  = "zc:ens10f1";
  char *device_10_out = "zc:ens10f0";

  
  //outputyo.open("outputyo",ios::out|ios::trunc);
  outputyo.open("outputyo",ios::out);
  outputyoGPU.open("outputyoGPU",ios::out);
  //int cluster_id = DEFAULT_CLUSTER_ID+9; 
  u_int numCPU = sysconf( _SC_NPROCESSORS_ONLN );
  //pattern
  ifstream inFile;
  string line;
  //mwm
  int nocase=1, npats=0;

  cout << "numCPU: " << numCPU << endl;

  //dir[0].bind_core = dir[1].bind_core = -1;
    for(int i=0;i<32;i++)
    {
    	dir[i].bind_core = -1;
    }

  startTime.tv_sec = 0;

	while((c = getopt(argc,argv,"abc:e:g:hi:j:n:N:o:s:t:r:fv")) != '?') {
		if((c == 255) || (c == -1)) break;

		switch(c) {
			case 'h':
			printHelp();
			break;
		case 'a':
			wait_for_packet = 0;
			break;
		case 'f':
			flush_packet = 1;
			break;
		case 'v':
			verbose = 1;
			break;
		case 'b':
			bidirectional = 1;
			break;
		case 'c':
			cluster_id = atoi(optarg);
			break;
		case 'i':
			device1 = strdup(optarg);
			break;
		case 'o':
			device2 = strdup(optarg);
			break;
		case 'g':
			bind_mask = strdup(optarg);
			break;
		case 'e':
			match_mode = atoi(optarg);
			break;
		case 'n':
			num_threads = atoi(optarg);
			break;
		case 'N':
			two_NIC = atoi(optarg);
			break;
		case 's':
			blocksNumper = atoi(optarg);
			printf("\n blocksNumper =%d  ",blocksNumper);
			break;
		case 't':
			threadsperBlock = atoi(optarg);
			printf("\n threadsperBlock = %d\n",threadsperBlock);
			break;
		case 'j':
			rec_pkt_size = atoi(optarg);
			break;
		case 'r':
			buffer_times = atof(optarg);
			break;
    }
  }
  
  if (device1 == NULL) printHelp();
  if (device2 == NULL) printHelp();
  if (cluster_id < 0)  printHelp();

  /*if(bind_mask != NULL) {
    char *id;
    if ((id = strtok(bind_mask, ":")) != NULL)
      dir[0].bind_core = atoi(id) % numCPU;
    if ((id = strtok(NULL, ":")) != NULL)
      dir[1].bind_core = atoi(id) % numCPU;
  }*/
  if(bind_mask != NULL)
  {
    char *id = strtok(bind_mask, ":");
    int idx = 0;
	cout<<"id: ";
    while(id != NULL)
    {
      dir[idx++].bind_core = atoi(id) % numCPU;
	  cout << atoi(id) <<" ";
      if(idx >= num_threads) break;
      id = strtok(NULL, ":");
    }
	cout<<endl;
  }
  	CPU_Count = new int[num_threads*2];
	if(CPU_Count == NULL)
	{
		cout << "Error memory set for CPU_Count" << endl;
	}
	for(int i=0;i<(num_threads*2);i++)
	{
		CPU_Count[i] = 0;
	}
	///// 12/06 for match_mode 4 GPU-AC //1227 test time of bytes //20200820 for corexbuffer bbb
	buffer_size = blocksNumper*threadsperBlock*buffer_times;
		cout<<"buffer_size = " << buffer_size << endl;
	//bytes = sizeof(u_char)*buffer_size*(payloadlen+3);
	bytes = sizeof(u_char)*buffer_size*buffershift;
		cout<<"bytes = "<<bytes<<endl;
	
	nfound_GPU1=(int*)malloc(sizeof(int)*blocksNumper*threadsperBlock*buffer_times);
	nfound_GPU2=(int*)malloc(sizeof(int)*blocksNumper*threadsperBlock*buffer_times);
	//whoami_GPU malloc
	whoami_GPU =(int*)malloc(sizeof(int)*blocksNumper*threadsperBlock*buffer_times); //

	Tablehits1=(bool*)malloc(sizeof(bool)*blocksNumper*threadsperBlock*buffer_times);
	Tablehits2=(bool*)malloc(sizeof(bool)*blocksNumper*threadsperBlock*buffer_times);
	hipMalloc((void**)&d_nfound_GPU1,sizeof(int)*blocksNumper*threadsperBlock*buffer_times);
	hipMalloc((void**)&d_nfound_GPU2,sizeof(int)*blocksNumper*threadsperBlock*buffer_times);
	//d_whoami_GPU malloc
	hipMalloc((void**)&d_whoami_GPU ,sizeof(int)*blocksNumper*threadsperBlock*buffer_times); //

	hipMalloc((void**)&d_Tablehits1,sizeof(bool)*blocksNumper*threadsperBlock*buffer_times);
	hipMalloc((void**)&d_Tablehits2,sizeof(bool)*blocksNumper*threadsperBlock*buffer_times);
	hipMalloc((void**)&d_T1_char,65536/8);
	hipMalloc((void**)&d_T2_char,53760/8);
	hipMalloc((void**)&d_BT_char,1024);
	memset(nfound_GPU1,0,sizeof(int)*blocksNumper*threadsperBlock*buffer_times);
	memset(nfound_GPU2,0,sizeof(int)*blocksNumper*threadsperBlock*buffer_times);
	//whoami_GPU memset
	memset(whoami_GPU ,0,sizeof(int)*blocksNumper*threadsperBlock*buffer_times); //
	memset(Tablehits1,0,sizeof(bool)*blocksNumper*threadsperBlock*buffer_times);

	memset(Tablehits2,0,sizeof(bool)*blocksNumper*threadsperBlock*buffer_times);
	hipMemset(d_nfound_GPU1,0,sizeof(int)*blocksNumper*threadsperBlock*buffer_times);
	hipMemset(d_nfound_GPU2,0,sizeof(int)*blocksNumper*threadsperBlock*buffer_times);
	//d_whoami_GPU memset
	hipMemset(d_whoami_GPU ,0,sizeof(int)*blocksNumper*threadsperBlock*buffer_times); //

	for(int i=0;i<8;i++) //bitmasks 128, 64, 32, 16, 8, 4, 2, 1
	{
		bitmasks[i] = pow(2,7-i);
		//		printf("%d :: bitmask = %u \n", i ,bitmasks[i]);
	}
	
	//*nfound_GPU=0;
	sem_init(&os_sem,0,1);
	int* partcial_sum;
	//比對所需之宣告
	hipHostMalloc((void**)&Central_memory,bytes*3);//3

	hipHostMalloc((void**)&CPU_Central_memory,num_threads*bytes*2+31);
	//printf("Before CPU_Central_memory: %p, %d\n", CPU_Central_memory,intptr_t(CPU_Central_memory));
	CPU_Central_memory = CPU_Central_memory + (32-(intptr_t(CPU_Central_memory)%32));
	//printf("After CPU_Central_memory: %p, %d\n", CPU_Central_memory,intptr_t(CPU_Central_memory));

	//CPU_Central_memory = (u_char*)malloc(num_threads*bytes*2);

	hipMalloc((void**)&d_Central_memory,2*bytes); //1119 
	memset(Central_memory,0,bytes*3);
	memset(CPU_Central_memory,0,num_threads*bytes*2);
	
	hipMemset(d_Central_memory,0,2*bytes);
	
	Buffer_location = new u_char*[buffer_size]; // recording each startIdx in buffer
	for(int i=0;i<buffer_size;i++)
	{
		Buffer_location[i] = &*(CPU_Central_memory+(i*buffershift));
	}

	Central_memory_tail = Central_memory;
	first_buffer = Central_memory + bytes;
	second_buffer = Central_memory + 2*bytes;
	
	///////////////////////////////////  T1
	FILE *fptr;
	fptr = fopen( T1_FILE_PATH ,"r");	
	t1size = GetFileLength(fptr);
  

	printf("T1 size:%d \n", t1size);
	T1 = (unsigned char*)malloc(t1size*sizeof(char));
	if( T1 == NULL )
	{
		fprintf(stderr, "記憶體不足\n");
		exit(1);
	}//fgets( T1 , t1size , fptr );
	//write into mem
	for( i = 0 ; i < t1size ; i++ )
	{
		T1[i] = fgetc(fptr);
		
	}

	///////////////////////////////////// BT
	fptr = fopen( BT_FILE_PATH ,"r");
  
	btsize = GetFileLength(fptr);
	BT = (unsigned char*)malloc(btsize*sizeof(char));
	if( BT == NULL )
	{
		fprintf(stderr, "記憶體不足\n");
		exit(1);
	}
	for( i = 0 ; i < btsize ; i++ )
	{
		BT[i] = fgetc(fptr);
	}
	cout << "btsize: " << btsize << endl;

	fptr = fopen( T2_FILE_PATH ,"r");
	t2size = GetFileLength(fptr);
	T2 = (unsigned char*)malloc(t2size*sizeof(char));
	if( T2 == NULL )
	{
		fprintf(stderr, "記憶體不足\n");
		exit(1);
	}
	for( i = 0 ; i < t2size ; i++ )
	{
		T2[i] = fgetc(fptr);
	}
  
  
	fclose(fptr);
	win_size = 65536/btsize;
	printf("win_size = %d   btsize = %d  t2size = %d t1size = %d \n",win_size, btsize,t2size,t1size);
	
	///////////////////////////////////  T1_bool
	T1_bool =(bool*)malloc(256*256*sizeof(bool));
	for(int c_1 = 0; c_1< 255; c_1++)
	{
		for( int c_2 = 0 ; c_2 < 255;  c_2++)
		{
			int charIndex = c_1*255+c_2;
			T1_bool[charIndex]= T1[ charIndex*4/8] & (bitmasks[(charIndex*4)%8+3]);
		}
	}
	
	/////////////////////////////////// T1b_bool
	
	fptr = fopen( T1b_FILE_PATH ,"r");
	t1bsize = GetFileLength(fptr);
	
	cout << "t1bsize: " << t1bsize << endl;
	
	T1b = (unsigned char*)malloc(t1bsize*sizeof(u_char));
	if( T1b == NULL )
	{
		fprintf(stderr, "記憶體不足\n");
		exit(1);
	}
	for (int i=0;i<t1bsize;i++) //655360 = 256*256
	{
		T1b[i] = fgetc(fptr);
		//ss << T2b[i];
		//ss >> boolalpha  >> T2_bool[i];
		T1b_bool[i] = T1b[i]&1;
	}
	
	fclose(fptr);

	/////////20190605 for T1_char

	fptr = fopen( T1c_FILE_PATH , "r");
	t1csize = GetFileLength(fptr);

	cout << "t1csize: " << t1csize << endl;
	T1c = (unsigned char*)malloc(t1csize*sizeof(u_char));
	if( T1c == NULL )
	{
		fprintf(stderr, "記憶體不足\n");
		exit(1);
	}
	for(int i=0;i<t1csize;i++) //8192
	{
		T1_char[i] = fgetc(fptr); // T1T1

	}
	
	for(int i=0;i<255;i++)
	{
		if( *(T1b_bool+i) != CheckT1(i) )
		{
			cout << "Different!!! at: " << i  << endl;
		}
		if( ((T1[i]>>4)&1) != CheckT1(i*2))
		{
			cout << "T1 is Different at: " << i*2 << endl;
		}
		if(((T1[ i*4/8])&(bitmasks[(i*4)%8+3])) != CheckT1(i) && ((T1[ i*4/8])&(bitmasks[(i*4)%8+3])) != 16 )
		{
			cout << "T1 is Different at: " << i << endl;
		}
	}
	cout << "T1_table_sc: " << endl;
	for(int i=0;i<10;i++)
	{
		cout << ((T1[i]>>4)&1) << ", " << (T1[i]&1) << ", " ;
	}
	cout << endl;

	fclose(fptr);

	/////////20190718 for T2_char

	fptr = fopen( T2c_FILE_PATH , "r");
	t2csize = GetFileLength(fptr);

	cout << "t2csize: " << t2csize << endl;
	T2c = (unsigned char*) malloc(t2csize*sizeof(u_char));
	if( T2c == NULL )
	{
		fprintf(stderr, "記憶體不足\n");
		exit(1);
	}
	for(int i=0;i<t2csize;i++) //6720
	{
		T2_char[i] = fgetc(fptr); // T2T2
	}
	cout << "Test T2_char: " << endl;
	for(int j=0;j<10;j++)
	{
		for(int i=0;i<8;i++)
		{
			cout <<  ((T2_char[j]>>(7-i))&1);
		}
	}
	cout << endl;

	fclose(fptr);


	
	/////////20190315 for popcount
	unsigned long long int numof64;
	numof64 = BitArrayToInt(T1_bool,11840,11891);
	cout << "numof64: " << numof64 << endl;
	cout << "popcount_3: " << popcount_3(numof64) << endl;
	
	fptr = fopen( T2b_FILE_PATH ,"r");
	t2bsize = GetFileLength(fptr);

	cout << "t2bsize: " << t2bsize << endl;
	
	//stringstream T2ss;
	T2b = (unsigned char*)malloc(t2bsize*sizeof(u_char));
	if( T2b == NULL )
	{
		fprintf(stderr, "記憶體不足\n");
		exit(1);
	}
	for (int i=0;i<t2bsize;i++)
	{
		T2b[i] = fgetc(fptr);
		//ss << T2b[i];
		//ss >> boolalpha  >> T2_bool[i];
		T2_bool[i] = T2b[i]&1;
	}
	for(int i=0;i<255;i++)
	{
		if(T2_bool[i]!=CheckT2(i))
		{
			cout << "T2 Different!!! at: " << i << endl;
		}
	}

  /////////////////////////////////////  build fullTable
	int charIndex = 0;
	int charIndex_winsize = 0;
	//int local_sum =0;
	int t2Loc = 0;
	int counterTest =0 ;
	//r = udaMallocManaged(&(fullTable),(65536*128));//2^16 *  2^7
	//r = hipMallocManaged(&(d_fullTable),sizeof(u_char)*65536*32);
	
	fullTable = (u_char*)malloc(65536*32);
	for(int c_1 = 0; c_1< 256; c_1++){
		for( int c_2 = 0 ; c_2 < 256;  c_2++){
			charIndex = c_1*256+c_2;
			charIndex_winsize=	charIndex-(charIndex%win_size); //貌似沒用
			//local_sum = 0;
			
			if(T1[ charIndex*4/8] & (bitmasks[(charIndex*4)%8+3]) )
			{
				for(int i = 0 ; i <32; i++)
				{	
						fullTable[charIndex*32+i]=T2[t2Loc*32+i];
						//fullTable[charIndex*32+i]=T2_char[t2Loc*32+i];
						counterTest++;
				}
				T2ptr[charIndex] = &T2[t2Loc*32];	
				t2Loc++;
				//t2Loc = BT[charIndex/win_size]+local_sum;
				//T2_bool[charIndex] = &T2b[t2Loc*32];
			}else{
				for(int i = 0 ; i <32; i++)
				{	
					fullTable[charIndex*32+i]=0;
				}	
				T2ptr[charIndex] =NULL;
			}		
		}
	}
	
	for(int i = 0 ; i<65536 ; i++)
	{
		if(T2ptr[i])
		{
			t2Loc--;
		}			
	}
	
	cout<<"counterTest  "<<counterTest<<endl<<"t2Loc  "<<t2Loc<<endl;
	
	hipMalloc((void**)&d_fullTable,sizeof(u_char)*65536*32);
	
	hipMemcpy(d_fullTable,fullTable,65536*32,hipMemcpyHostToDevice);
	
	hipBindTexture(0,texFulltable,d_fullTable ,65536*32);
	hipBindTexture(NULL,texacsm,acsm,sizeof(acsm));
	//free(fullTable);
  
  //////////////////////////////////////////////////////////////////////// build acsm(AC)
  //acsm pattern
  vector<string> patNumstr;
  vector<string> patTERN;
  cout<<"Start to read Pattern file!"<<endl;
  
  //load pattern data
  inFile.open(PATTERN_FILE_PATH);
  while(getline(inFile,line))
  {
  	patTERN.push_back(line);
  }
  inFile.close();
  inFile.clear();
  cout<<"End read Pattern file!"<<endl;
  //end!

  //load pattern num data
  inFile.open(PAT_FILE_PATH);
  while(getline(inFile,line))
  {
  	patNumstr.push_back(line);
  }
  inFile.close();
  inFile.clear();
  cout<<"Done for load pattern!"<<endl;

  //stable_sort(patNumstr.begin(),patNumstr.end(), sortRule);

  vector<int> patternNum[patNumstr.size()];
  for(int i=0; i<patNumstr.size();i++)
  {
  	for(int j=0;j<patNumstr[i].size();j=j+3)
	{
		patternNum[i].push_back(StrToInt(patNumstr[i].substr(j,3)));
	}
  }
  //end!
  
  //snort mwm
  ps = mwmNew();
  //snort ac
#define acsm_print printf ("MAX_Memory: %d bytes, acsmMacStates: %d, acsmNumStates: %d  mem: acsm: %d maxstate: %d numstate: %d pattern %d statetalbe %d\n", max_memory, acsm->acsmMaxStates, acsm->acsmNumStates, &acsm, &(acsm->acsmMaxStates), &(acsm->acsmNumStates), &(acsm->acsmPatterns), &(acsm-<acsmStateTable));

  //acsmNew()
  printf("hipMallocManaged\n");
  r = hipMallocManaged(&(acsm), sizeof(ACSM_STRUCT));
  //err acsm_print printf("acsm NER\n");
  init_xlatcase();
  
  memset(acsm, 0, sizeof(ACSM_STRUCT));
  //end!

  //add pattern
  cout<<"add pattern"<<endl;
  for(int x=0; x<patNumstr.size();x++)
  {
  	char* s_mwm = (char*) malloc (sizeof(char)*(patternNum[x].size()+1));

	for(int y=0; y<patternNum[x].size();y++)
	{
		s_mwm[y] = (unsigned char)patternNum[x][y];
	}

	mwmAddPatternEx(ps, (unsigned char*)s_mwm, patternNum[x].size(), nocase, 0, 0, (void*)npats, 3000);
	acsmAddPattern(acsm, (unsigned char*)s_mwm, patternNum[x].size(), nocase, 0, 0, s_mwm, x);

	#ifdef _test
	patArray[npats] = s_mwm;
	#endif
	npats++;
  }
  mwmPrepPatterns(ps);
  
  cout<<"patNumstr.size() :"<<patNumstr.size()<<endl;

  //acsmCompile()
  cout<<"acsmCompile"<<endl;
  acsmCompile(acsm);

  ///////////////////////////////////////////////////////////////// pfring_zc info
  zc = pfring_zc_create_cluster(
    cluster_id, 
    max_packet_len(device1), 
    0, 
    ((2 * MAX_CARD_SLOTS) + 1) * (1 + bidirectional),
    NULL, //pfring_zc_numa_get_cpu_node(dir[0].bind_core)
    NULL /* auto hugetlb mountpoint */ 
  );

  if(zc == NULL) {
    fprintf(stderr, "pfring_zc_create_cluster error [%s] Please check your hugetlb configuration\n",
	    strerror(errno));
    return -1;
  }
	
	if(two_NIC == 0) //only use one NIC and one thread
	{
		for(int i = 0; i < num_threads; i++)
		{
			ss << i;
			ss >> idder;
			string temp1 = string(device1) + string(device_at) + idder; //ens4f1 + @ + i
			string temp2 = string(device2) + string(device_at) + idder; //ens4f0 + @ + i
			device1_rss[i]=strdup(temp1.c_str());
			device2_rss[i]=strdup(temp2.c_str());
			/*if (init_direction(&dir[0], device1, device2) < 0) 
				return -1;*/
			if (init_direction(&dir[i], device1_rss[i], device2_rss[i]) < 0)
			{
				return -1;
			}
			ss.clear();	
		}
	}else if(two_NIC > 0) //use both two NIC and 2*threads
	{
		for(int i = 0; i < num_threads/2; i++)
		{
			cout<<"i: "<<i<<endl;
			ss << i;
			ss >> idder;
			string temp1 = string(device1) + string(device_at) + idder;       //zc:ens4f1 + @ + i
			string temp2 = string(device2) + string(device_at) + idder;       //zc:ens4f0 + @ + i
			string temp3 = string(device_10_in) + string(device_at) + idder;  //zc:ens10f1 + @ + i
			string temp4 = string(device_10_out) + string(device_at) + idder; //zc:ens10f0 + @ + i
			
			device1_rss[i] = strdup(temp1.c_str());        //zc:ens4f1@i
			device2_rss[i] = strdup(temp2.c_str());        //zc:ens4f0@i
			device1_rss_NIC[i] = strdup(temp3.c_str());  //zc:ens10f1@i
			device2_rss_NIC[i] = strdup(temp4.c_str());  //zc:ens10f0@i
			/*if (init_direction(&dir[0], device1, device2) < 0) 
				return -1;*/
			if (init_direction(&dir[2*i], device1_rss[i], device2_rss[i]) < 0)
			{
				return -1;
			}
			if (init_direction(&dir[(2*i)+1], device1_rss_NIC[i], device2_rss_NIC[i]) < 0)
			{
				return -1;
			}
			ss.clear();	
		}
	}
  
	if (bidirectional)
	{
		if (init_direction(&dir[1], device2, device1) < 0) 
		{
			return -1;
		}
	}
	
  	//signal(SIGINT, sigproc);
	//signal(SIGTERM, sigproc);
	//signal(SIGINT, sigproc);

	pthread_create(&persistentyo, NULL, persistentkernel, NULL);
  	
	cout << "The ZX cluster [id: " << cluster_id << "][num consumer threads: " << num_threads << "] is running..." <<endl;
	for(int i = 0; i < num_threads; i++)
	{
		if (two_NIC==0)
		{
			cout<<"Thread"<<i<<": ";
			cout<<"In_device: "<<device1<<"@"<<i<<"  ;  Out_device: "<<device2<<"@"<<i<<endl;
		}else if (two_NIC > 0)
		{
			cout<<"Thread"<<i<<": ";
			cout<<"In_device: "<<device1<<"@"<<i<<", "<<device_10_in<<"@"<<i<<" ; ";
			cout<<"Out_device: "<<device2<<"@"<<i<<", "<<device_10_out<<"@"<<i<<endl;
		}
		/*pthread_create(&dir[0].thread, NULL, packet_consumer_thread, (void *) &dir[0]);
		if (bidirectional) pthread_create(&dir[1].thread, NULL, packet_consumer_thread, (void *) &dir[1]);*/
		pthread_create(&dir[i].thread, NULL, packet_consumer_thread, (void *) &dir[i]);
	}
	
	//output+again
    	signal(SIGINT,  sigproc);
	signal(SIGTERM, sigproc);
	signal(SIGINT,  sigproc);

  if (!verbose) while (!do_shutdown) {
    //sleep(ALARM_SLEEP);
	  usleep(countrate*1000000); //10000=1s
	  gettimeofday(&startREAL,NULL);
    print_stats();
    	  gettimeofday(&endREAL,NULL);
	  diffREAL += 1000000*(endREAL.tv_sec-startREAL.tv_sec)+endREAL.tv_usec-startREAL.tv_usec;
	  cout << "print_stats time: " << diffREAL << endl;
	  diffREAL = 0;
	  
  }
  //no again
  //signal(SIGINT, sigproc);
	
	for(int i = 0; i < num_threads; i++)
	{
		/*pthread_join(dir[0].thread, NULL);
		if (bidirectional) pthread_join(dir[1].thread, NULL);*/
		pthread_join(dir[i].thread, NULL);
	}
	cout<<"hello?"<<endl;
	outputyo.close();
	outputyoGPU.close();

  sleep(1);

  pfring_zc_destroy_cluster(zc);
	sem_destroy(&os_sem);
  return 0;
}

